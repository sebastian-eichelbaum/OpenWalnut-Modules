#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------
//
// Project: OpenWalnut ( http://www.openwalnut.org )
//
// Copyright 2010 RRZK, University of Cologne
// Copyright 2009 OpenWalnut Community, BSV@Uni-Leipzig and CNCF@MPI-CBS
// For more information see http://www.openwalnut.org/copying
//
// This file is part of OpenWalnut.
//
// OpenWalnut is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// OpenWalnut is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with OpenWalnut. If not, see <http://www.gnu.org/licenses/>.
//
//---------------------------------------------------------------------------

#include <deque>

#include <hip/hip_runtime.h>

#include "WCheckCudaError.h"
#include "WCudaReduce.cu"
#include "WMDetTractClusteringCudaKernel.h"
#include "WProgressWrapper.h"

const int nthreads = 192;       // number of cuda threads
const int gridsize = 32;        // don't schedule too large tasks - watchdog timer will kill long-running ones
const int maxkernels = 4;       // maximum number of concurrent kernels,
                                // avoids possible problems because of too many pending kernels
const int allowedlength = 1024; // maximum allowable length of a fiber in __shared__ memory
const int warpsize = 32;        // you should not need to change this

const float LARGE_VALUE = 3.402823466e+38f; // FLT_MAX

/* reduction operation: sum */
struct ReduceOpSum
{
   static inline __device__ float eval(const float x, const float y)
   {
      return x + y;
   }
};
static inline __device__ void reducesum(volatile float *s, unsigned int n)
{
   Reduce<ReduceOpSum>::apply(s, n);
}


/* compute asymmetric metric from one fiber to another,
 * threshold2 is the square of the proximity threshold */
static __device__ float distasym( const int qsize, const float3 *q,
        const int rsize, const float3 *r,
        const float threshold2 )
{
    if( rsize <= 0 || qsize <= 0 )
        return 0.f;

    extern float __shared__ minaccum[];
    minaccum[threadIdx.x] = 0.;

    // for every point in the first fiber q...
    for( unsigned int i = threadIdx.x; i < qsize; i += blockDim.x )
    {
        // ...find the squared distance to closest point on fiber q
        float mind2 = LARGE_VALUE;
        for( unsigned int j = 0; j < rsize; ++j )
        {
            const float dx = q[i].x - r[j].x;
            const float dy = q[i].y - r[j].y;
            const float dz = q[i].z - r[j].z;

            const float d2 = dx*dx + dy*dy + dz*dz;
            mind2 = fminf( mind2, d2 );
        }

        if( mind2 > threshold2 )
            minaccum[threadIdx.x] += sqrtf( mind2 );
    }

    // add closest distances for all points on r
    __syncthreads();
    reducesum( minaccum, min(qsize, blockDim.x) );

    return minaccum[0] / qsize;
}

/**
 * copy a fiber from global to __shared__ memory
 */
static __device__ void loadFiber( float3 *x, const float *coords, const int start, const int length )
{
    for( unsigned int i = threadIdx.x; i < length; i += blockDim.x )
    {
        x[i].x = coords[(start+i)*3];
        x[i].y = coords[(start+i)*3 + 1];
        x[i].z = coords[(start+i)*3 + 2];
    }
}

/**
 * compute asymmetric Zhang metric for fibers with numbers corresponding to thread block grid coordinate 
 *
 * nshmfibs:     number of fibers to copy to __shared__ memory (have to be short enough)
 * maxlength:    maximum length of a fiber to be loaded into __shared__ memory
 * distmat:      output distance matrix (ntracts x ntracts)
 * tilex, tiley: ntracts x ntracts fiber array has been split into tiles,
 *               starting coordinates of fiber tile to be processed by this kernel call
 * coords:       interleaved (x[0], y[0], z[0], x[0], ...) coordinate array
 * ntracts:      total no. of fibers
 * offsets:      start indices of fibers
 * lengths:      fiber lengths
 * threshold2:   ignore points closer than the square root of this distance
 */
template< int nshmfibs, int maxlength >
static __global__ void distKernel( float *distmat,
        const size_t tilex, const size_t tiley,
        const float *coords,
        const int ntracts, const int *offsets, const int *lengths,
        const float threshold2 )
{
    float3 __shared__ q[nshmfibs > 1 ? maxlength : 0];
    float3 __shared__ r[nshmfibs > 0 ? maxlength : 0]; // store fiber coordinates

    // compute fiber index for this thread block
    const int qidx = tilex + blockIdx.x;
    const int ridx = tiley + blockIdx.y;
    const int qsize = lengths[qidx];
    const int rsize = lengths[ridx];

    // copy fibers to shared memory
    if( nshmfibs > 1 )
        loadFiber( q, coords, offsets[qidx], qsize );
    if( nshmfibs > 0 )
    {
        loadFiber( r, coords, offsets[ridx], rsize );
        __syncthreads();
    }

    // compute both asymmetric distances
    const float distqr =
        nshmfibs > 1
        ? distasym( qsize, q, rsize, r, threshold2 )
        : nshmfibs > 0
          ? distasym( qsize, (float3 *)&coords[offsets[qidx]*3], rsize, r, threshold2 )
          : distasym( qsize, (float3 *)&coords[offsets[qidx]*3],
                  rsize, (float3 *)&coords[offsets[ridx]*3], threshold2 );

    if( threadIdx.x == 0 )
        distmat[qidx*ntracts + ridx] = distqr;
}

bool distCuda(float *h_distmat,
        const int ncoords, const float *h_coords,
        const int ntracts, const int *h_offsets, const int *h_lengths,
        const float threshold,
        ProgressWrapper *progress )
{
    // allocate memory on device
    float *d_coords = NULL;
    int *d_lengths = NULL;
    int *d_offsets = NULL;
    float *d_distmat = NULL;

    hipMalloc( &d_coords, sizeof( float ) * ncoords * 3 );
    hipMalloc( &d_lengths, sizeof( int ) * ntracts );
    hipMalloc( &d_offsets, sizeof( int ) * ntracts );
    hipMalloc( &d_distmat, sizeof( float ) * ntracts * ntracts );
    hipMemset( d_distmat, 0, sizeof( float ) * ntracts * ntracts );

    // copy data to device
    hipMemcpy( d_coords, h_coords, sizeof( float ) * ncoords * 3, hipMemcpyHostToDevice );
    hipMemcpy( d_lengths, h_lengths, sizeof( int ) * ntracts, hipMemcpyHostToDevice );
    hipMemcpy( d_offsets, h_offsets, sizeof( int ) * ntracts, hipMemcpyHostToDevice );

    // number of kernel calls (corresponding to "tiles" from tract pair matrix),
    const int numBlocks = (ntracts+gridsize-1)/gridsize * (ntracts+gridsize-1)/gridsize;
    progress->start( "Tract distances", numBlocks );
    std::deque<hipEvent_t> kernels; // store scheduled, but not yet terminated kernels

    bool success = true;
    for( int i = 0; i < ntracts; i += gridsize )
    {
        for( int j = 0; j < ntracts; j += gridsize )
        {
            dim3 grid(std::min(gridsize, ntracts-i), std::min(gridsize, ntracts-j));

            hipEvent_t kernel;
            kernels.push_back( kernel );
            hipEventCreate( &kernels.back() );
            hipEventRecord( kernels.back(), 0 );

#define DISTKERNEL( nshmfibs, maxlength ) \
            distKernel< nshmfibs, maxlength > \
            <<< grid, min(nthreads,maxlength), sizeof(float)*min(nthreads,maxlength) >>> \
                ( d_distmat, \
                  i, j, \
                  d_coords, \
                  ntracts, d_offsets, d_lengths, \
                  threshold*threshold )

            if( max( h_lengths[i], h_lengths[j] ) <= warpsize )
            {
                DISTKERNEL( 2, warpsize );
            }
            else if( h_lengths[j] <= allowedlength )
            {
                if( h_lengths[j] <= 32 )
                    DISTKERNEL( 1, 32 );
                else if( h_lengths[j] <= 64 )
                    DISTKERNEL( 1, 64 );
                else if( h_lengths[j] <= 128 )
                    DISTKERNEL( 1, 128 );
                else if( h_lengths[j] <= 256 )
                    DISTKERNEL( 1, 256 );
                else if( h_lengths[j] <= 512 )
                    DISTKERNEL( 1, 512 );
                else
                    DISTKERNEL( 1, allowedlength );
            }
            else
            {
                DISTKERNEL( 0, 0 );
            }
            if( !checkCudaError(&success, hipGetLastError()) )
                break;

            if( kernels.size() > maxkernels )
            {
                hipEventSynchronize( kernels.front() );
                if( !checkCudaError(&success, hipGetLastError()) )
                    break;

                kernels.pop_front();
                ++*progress;
            }
        }

        if( !success )
            break;
    }

    // wait for all kernels to finish
    while( success && !kernels.empty() )
    {
        hipEventSynchronize( kernels.front() );
        if( !checkCudaError(&success, hipGetLastError()) )
            break;

        kernels.pop_front();
        ++*progress;
    }
    hipDeviceSynchronize();
    checkCudaError(&success, hipGetLastError());

    // copy result data back to host
    hipMemcpy( h_distmat, d_distmat, sizeof( float ) * ntracts * ntracts, hipMemcpyDeviceToHost );

    hipFree( d_coords );
    hipFree( d_distmat );
    hipFree( d_lengths );
    hipFree( d_offsets );

    progress->finish();

    return success;
}
